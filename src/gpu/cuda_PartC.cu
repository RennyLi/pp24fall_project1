#include "hip/hip_runtime.h"
//
// Created by Liu Yuxuan on 2024/9/11
// Modified from Zhong Yebin's PartB on 2023/9/16
//
// Email: yebinzhong@link.cuhk.edu.cn
// Email: yuxuanliu1@link.cuhk.edu.cn
//
// CUDA implementation of bilateral filtering on JPEG image
//

#include <iostream>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>
#include "../utils.hpp"

__constant__ float d_sigma_s = 15.0f;
__constant__ float d_sigma_r = 30.0f;

__device__ unsigned char d_clamp_pixel_value(float pixel)
{
    return pixel > 255 ? 255
           : pixel < 0 ? 0
                       : static_cast<unsigned char>(pixel);
}

// kernel function for applying bilateral filtering on an image channel
__global__ void bilateral_filter_kernel(const ColorValue* input_channel, ColorValue* output_channel,
                                        int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 1 && x < width - 1 && y >= 1 && y < height - 1) {
        int idx = y * width + x;

        float sum = 0.0f;
        float norm_factor = 0.0f;
        float center_value = input_channel[idx];

        // loop through the 3*3 kernel
        for (int ky = -1; ky <= 1; ky++) {
            for (int kx = -1; kx <= 1; kx++) {
                int neighbor_x = x + kx;
                int neighbor_y = y + ky;
                int neighbor_idx = neighbor_y * width + neighbor_x;

                float spatial_dist = kx * kx + ky * ky;
                float spatial_weight = expf(-spatial_dist / (2 * d_sigma_s * d_sigma_s));

                float range_dist = center_value - input_channel[neighbor_idx];
                float range_weight = expf(-(range_dist * range_dist) / (2 * d_sigma_r * d_sigma_r));

                float weight = spatial_weight * range_weight;
                sum += input_channel[neighbor_idx] * weight;
                norm_factor += weight;
            }
        }

        output_channel[idx] = d_clamp_pixel_value(sum / norm_factor);
    }
}

// utility function to initiate the CUDA kernel
void apply_bilateral_filter_cuda(const JpegSOA& input_jpeg, JpegSOA& output_jpeg, int width, int height) {
    // reserve memory on the gpu for the input and output channels
    ColorValue *d_r_input, *d_g_input, *d_b_input;
    ColorValue *d_r_output, *d_g_output, *d_b_output;

    size_t channel_size = width * height * sizeof(ColorValue);
    hipMalloc((void**)&d_r_input, channel_size);
    hipMalloc((void**)&d_g_input, channel_size);
    hipMalloc((void**)&d_b_input, channel_size);
    hipMalloc((void**)&d_r_output, channel_size);
    hipMalloc((void**)&d_g_output, channel_size);
    hipMalloc((void**)&d_b_output, channel_size);

    // transfer inpit data from the host to the device
    hipMemcpy(d_r_input, input_jpeg.r_values, channel_size, hipMemcpyHostToDevice);
    hipMemcpy(d_g_input, input_jpeg.g_values, channel_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b_input, input_jpeg.b_values, channel_size, hipMemcpyHostToDevice);

    // configure the dimensions of the cuda grid and blocks
    dim3 blockDim(32, 32);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    // initiate the cuda kernel for each channel
    bilateral_filter_kernel<<<gridDim, blockDim>>>(d_r_input, d_r_output, width, height);
    bilateral_filter_kernel<<<gridDim, blockDim>>>(d_g_input, d_g_output, width, height);
    bilateral_filter_kernel<<<gridDim, blockDim>>>(d_b_input, d_b_output, width, height);

    // transfer the output data back to the host
    hipMemcpy(output_jpeg.r_values, d_r_output, channel_size, hipMemcpyDeviceToHost);
    hipMemcpy(output_jpeg.g_values, d_g_output, channel_size, hipMemcpyDeviceToHost);
    hipMemcpy(output_jpeg.b_values, d_b_output, channel_size, hipMemcpyDeviceToHost);

    // release the memory on the device
    hipFree(d_r_input);
    hipFree(d_g_input);
    hipFree(d_b_input);
    hipFree(d_r_output);
    hipFree(d_g_output);
    hipFree(d_b_output);
}

int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Invalid argument, should be: ./executable /path/to/input/jpeg /path/to/output/jpeg\n";
        return -1;
    }

    // load the input jpeg image in a structure-of-arrays format
    const char* input_filename = argv[1];
    const char* output_filename = argv[2];
    std::cout << "Input file from: " << input_filename << "\n";

    JpegSOA input_jpeg = read_jpeg_soa(input_filename);
    if (input_jpeg.r_values == nullptr) {
        std::cerr << "Failed to read input JPEG image\n";
        return -1;
    }

    // reserve memory for the output image
    JpegSOA output_jpeg;
    output_jpeg.width = input_jpeg.width;
    output_jpeg.height = input_jpeg.height;
    output_jpeg.num_channels = input_jpeg.num_channels;
    output_jpeg.color_space = input_jpeg.color_space;
    output_jpeg.r_values = new ColorValue[output_jpeg.width * output_jpeg.height];
    output_jpeg.g_values = new ColorValue[output_jpeg.width * output_jpeg.height];
    output_jpeg.b_values = new ColorValue[output_jpeg.width * output_jpeg.height];

    hipEvent_t start, stop;
    float gpuDuration;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);  

    // utilize cuda to perform bilateral filtering
    apply_bilateral_filter_cuda(input_jpeg, output_jpeg, input_jpeg.width, input_jpeg.height);

    hipEventRecord(stop, 0);  
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuDuration, start, stop);
    std::cout << "GPU Execution Time: " << gpuDuration << " milliseconds" << std::endl;

    std::cout << "Output file to: " << output_filename << "\n";
    if (export_jpeg(output_jpeg, output_filename)) {
        std::cerr << "Failed to write output JPEG\n";
        return -1;
    }

    delete[] input_jpeg.r_values;
    delete[] input_jpeg.g_values;
    delete[] input_jpeg.b_values;
    delete[] output_jpeg.r_values;
    delete[] output_jpeg.g_values;
    delete[] output_jpeg.b_values;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}